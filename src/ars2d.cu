#include "hip/hip_runtime.h"
/**
 * CudARS: Angular Radon Spectrum - CUDA version
 * Copyright (C) 2017-2020 Dario Lodi Rizzini.
 * Copyright (C) 2021- Dario Lodi Rizzini, Ernesto Fontana.
 *
 * CudARS is free software: you can redistribute it and/or modify
 * it under the terms of the GNU Lesser General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * CudARS is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU Lesser General Public License for more details.
 *
 * You should have received a copy of the GNU Lesser General Public License
 * along with CudARS.  If not, see <http://www.gnu.org/licenses/>.
 */

#include "cudars/ars2d.cuh"

// --------------------------------------------------------
// DIVISION IN CHUNKS (FOR BIG IMAGES)
// --------------------------------------------------------

//__host__

int numChunks(int totNumPts, int chunkSz)
{
    if (totNumPts % chunkSz > 1024)
        return (totNumPts / chunkSz) + 1;
    else
        return max(1, totNumPts / chunkSz);
}

//__host__

thrust::pair<int, int> chunkStartEndIndices(int round, int totNumPts, int chunkSz)
{
    thrust::pair<int, int> pr;

    // TODO: adaptation for cases when last chunk is small (< ~1000 pts)
    if (totNumPts <= chunkSz)
    {
        pr.first = 0;
        pr.second = totNumPts - 1;

        return pr;
    }
    pr.first = chunkSz * round;
    pr.second = min(totNumPts - 1, chunkSz * (round + 1) - 1);

    if (totNumPts - pr.second < 1025)
        pr.second = totNumPts - 1;

    return pr;
}

// --------------------------------------------------------
// 2D->1D INDICIZATION IN FOURIER COEFFICIENT MATRIX
// --------------------------------------------------------

__device__ int getIfromTid(int tid, int n)
{
    if (tid < 0 || n < 0)
        return -1;
    const int nId = n - 1; // max n in ids (indices start from 0)
    //    const int tidStart = tid; //useful for debugging
    /*i is equal to the number of times that we can subtract NID, NID-1, NID-2, ...
     from tid before tid goes below 0*/
    int i = 0;
    if (tid < nId)
    {
        //        printf("tid %d i %d       n %d\n", tidStart, i, n);
        return i;
    }

    if (tid >= 0.5 * nId * (nId + 1))
    { // maybe change this to improve efficiency?
        //        printf("tid %d i %d       n %d\n", tidStart, i, n);
        i = n;
        return n;
    }

    while (tid >= 0)
    {
        tid -= (nId - i);
        i++;
    }
    //    printf("tid %d i %d       n %d\n", tidStart, i - 1, n);
    return i - 1;
}

__device__ int getJfromTid(int tid, int n, int i)
{
    if (tid < 0 || n < 0 || i < 0)
        return -1;

    const int nId = n - 1; // max n in ids (indices start from 0)
    //    const int tidStart = tid; //useful for debugging
    //    const int iStart = i; //useful for debugging
    /*i is equal to the number of times that we can subtract NID, NID-1, NID-2, ...
     from tid before tid goes below 0*/
    int j = -1;

    if (tid >= 0.5 * nId * (nId + 1) || i > nId)
    { // maybe change this to improve efficiency?
        j = n;
        //        printf("tid %d i %d j %d       n %d\n", tidStart, iStart, j, n);
        return n;
    }

    while (i > 0)
    {
        tid -= (nId - i);
        i--;
    }
    j = tid + 1;
    //    printf("tid %d i %d j %d       n %d\n", tidStart, iStart, j, n);
    return j;
}

// --------------------------------------------------------
// PNEBI FUNCTIONS
// --------------------------------------------------------

__device__ double evaluatePnebi0Polynom(double x)
{
    double t, t2, tinv, val;

    if (x < 0.0)
        x = -x;
    t = x / 3.75;

    if (t < 1.0)
    {
        t2 = t * t;
        val = 1.0 + t2 * (3.5156229 + t2 * (3.0899424 + t2 * (1.2067492 + t2 * (0.2659732 + t2 * (0.360768e-1 + t2 * 0.45813e-2)))));
        val = 2.0 * exp(-x) * val;
    }
    else
    {
        tinv = 1 / t;
        val = (0.39894228 + tinv * (0.1328592e-1 + tinv * (0.225319e-2 + tinv * (-0.157565e-2 + tinv *
                                                                                                    (0.916281e-2 + tinv * (-0.2057706e-1 + tinv * (0.2635537e-1 + tinv * (-0.1647633e-1 + tinv * 0.392377e-2))))))));
        val = 2.0 * val / sqrt(x);
    }

    return val;
}

__device__ void evaluatePnebiVectorGPU(int n, double x, double *pnebis, int pnebisSz)
{
    double factor, seqPrev, seqCurr, seqNext;
    //    if (pnebis.size() < n + 1) { //questa condizione dovrebbe essere già garantita prima della chiamata di evaluatePnebiVectorGPU
    //        pnebis.resize(n + 1); //ovvero: il questo resizing non dovrebbe essere necessario
    //    }

    if (x < 0.0)
        x = -x;

    // If x~=0, then BesselI(0,x) = 1.0 and BesselI(k,x) = 0.0 for k > 0.
    // Thus, PNEBI(0,x) = 2.0 and PNEBI(k,x) = 0.0 for k > 0.
    if (x < 1e-6)
    {
        pnebis[0] = 2.0;
        for (int i = 1; i < pnebisSz; ++i)
            pnebis[i] = 0.0;
        return;
    }

    // Computes bessel function using back recursion
    factor = 2.0 / x;
    seqPrev = 0.0; // bip
    seqCurr = 1.0; // bi
    seqNext = 0.0; // bim
    for (int k = 2 * (n + (int)sqrt(40.0 * n)); k >= 0; --k)
    {
        seqNext = seqPrev + factor * k * seqCurr;
        seqPrev = seqCurr;
        seqCurr = seqNext;
        if (k <= n)
        {
            pnebis[k] = seqPrev;
        }
        // To avoid overflow!
        if (seqCurr > cudars::BIG_NUM)
        {
            seqPrev *= cudars::SMALL_NUM;
            seqCurr *= cudars::SMALL_NUM;
            for (int i = 0; i < pnebisSz; ++i)
            {
                pnebis[i] *= cudars::SMALL_NUM;
            }
            // std::cerr << __FILE__ << "," << __LINE__ << ": ANTI-OVERFLOW!" << std::endl;
        }
    }

    double scaleFactor = evaluatePnebi0Polynom(x) / pnebis[0];
    for (int i = 0; i < pnebisSz; ++i)
    {
        pnebis[i] = scaleFactor * pnebis[i];
    }
}

// --------------------------------------------------------
// GLOBAL CUDA KERNELS
// --------------------------------------------------------

__global__ void iigDw(cudars::Vec2d *means, double sigma1, double sigma2, int numPts, int fourierOrder, int numColsPadded, cudars::ArsKernelIso2dComputeMode pnebiMode, double *coeffsMat)
{
    //    a.insertIsotropicGaussians(points, sigma);

    int index = blockIdx.x * blockDim.x + threadIdx.x; // index runs through a single block
    int stride = blockDim.x * gridDim.x;               // total number of threads in the grid

    const int totalNumComparisons = gridDim.x * blockDim.x;

    for (int tid = index; tid < totalNumComparisons; tid += stride)
    {

        int i = getIfromTid(tid, numPts);
        int j = getJfromTid(tid, numPts, i);
        //        printf("i %d j %d\n", i, j);

        if (i >= numPts || j >= numPts || j <= i)
        {
            //            printf("BAD INDEXING!!!!\n"); //could actually be ok because of padding... maybe optimization can be further improved in this regard
            continue;
        }

        cudars::Vec2d vecI = means[i];
        cudars::Vec2d vecJ = means[j];

        //            isotropicKer_.init(means[i], means[j], sigma);
        double dx, dy;
        dx = vecJ.x - vecI.x;
        dy = vecJ.y - vecI.y;
        double phi;

        //        if (dx == 0 && dy == 0) {
        //                        phi = 0.0; //mathematically undefined
        //            //            for (int k = 0; k <= numColsPadded; ++k) {
        //            //                int rowIndex = (i * numPtsAfterPadding) + j; //it's more a block index rather than row
        //            //                coeffsMat[rowIndex * numColsPadded + k] = 0.0;
        //            //            }
        ////            continue;
        //
        //        } else
        phi = atan2(dy, dx);

        double sigmaValSq = sigma1 * sigma1 + sigma2 * sigma2;
        double lambdaSqNorm = 0.25 * (dx * dx + dy * dy) / sigmaValSq;

        //            isotropicKer_.updateFourier(arsfOrder_, coeffs_, w);
        double wNorm = 1.0 / (numPts * numPts);
        double weight = wNorm / sqrt(2.0 * M_PI * sigmaValSq);

        // updating Fourier coefficients (2 modes)
        if (pnebiMode == cudars::ArsKernelIso2dComputeMode::PNEBI_DOWNWARD)
        {
            //                updateARSF2CoeffRecursDown(lambdaSqNorm, phi, w2, nFourier, coeffs);

            double cth2, sth2;
            cth2 = cos(2.0 * phi);
            sth2 = sin(2.0 * phi);
            //                updateARSF2CoeffRecursDown(lambda, cth2, sth2, factor, n, coeffs);

            int pnebisSz = fourierOrder + 1;
            // TODO: find a better solution instead of hard-coding 21
            double pnebis[21]; // Fourier Order + 1
            if (pnebis == nullptr)
                printf("ERROR ALLOCATING WITH NEW[]!\n");
            for (int pn = 0; pn < pnebisSz; ++pn)
                pnebis[pn] = 0.0;

            double sgn, cth, sth, ctmp, stmp;

            // Fourier Coefficients
            //                if (coeffs.size() != 2 * n + 2) {
            //                    std::cerr << __FILE__ << "," << __LINE__ << ": invalid size of Fourier coefficients vector " << coeffs.size() << " should be " << (2 * n + 2) << std::endl;
            //                    return;
            //                }

            evaluatePnebiVectorGPU(fourierOrder, lambdaSqNorm, pnebis, pnebisSz);
            //                ARS_PRINT(pnebis[0]);

            //!!!! factor = w2
            double factor = weight;
            int rowIndex = tid; // = tid
            coeffsMat[rowIndex * numColsPadded + 0] += 0.5 * factor * pnebis[0];
            //            printf("coeff 0 %f\n", 0.5 * factor * pnebis[0]);

            sgn = -1.0;
            cth = cth2;
            sth = sth2;
            //!!!! n in the for below is fourierOrder
            for (int k = 1; k <= fourierOrder; ++k)
            {
                //                printf("coeff %d %f\n", 2 * k, factor * pnebis[k] * sgn * cth);
                //                printf("coeff %d %f\n", 2 * k + 1, factor * pnebis[k] * sgn * sth);
                coeffsMat[(rowIndex * numColsPadded) + (2 * k)] += factor * pnebis[k] * sgn * cth;
                coeffsMat[(rowIndex * numColsPadded) + ((2 * k) + 1)] += factor * pnebis[k] * sgn * sth;
                sgn = -sgn;
                ctmp = cth2 * cth - sth2 * sth;
                stmp = sth2 * cth + cth2 * sth;
                cth = ctmp;
                sth = stmp;
            }

            //            delete pnebis;
        }
        else
            printf("ERROR: pnebi mode is NOT Downward!\n");
    }
}

__global__ void iigLut(cudars::Vec2d *means, double sigma1, double sigma2, int numPts, int numPtsAfterPadding, int fourierOrder, int numColsPadded, cudars::ArsKernelIso2dComputeMode pnebiMode, cudars::PnebiLUT &pnebiLUT, double *coeffsMat)
{
    //    a.insertIsotropicGaussians(points, sigma);

    int index = blockIdx.x * blockDim.x + threadIdx.x; // index runs through a single block
    int stride = blockDim.x * gridDim.x;               // total number of threads in the grid

    const int totalNumComparisons = numPtsAfterPadding * numPtsAfterPadding;

    for (int tid = index; tid < totalNumComparisons; tid += stride)
    {

        int j = tid % numPtsAfterPadding;
        int i = (tid - j) / numPtsAfterPadding;
        //        printf("i %d j %d\n", i, j);
        //        printf("tid %d i %d j %d tidIJ %d --- numPts %d numPtsAfterPadding %d numColsPadded %d totNumComp %d index %d\n", tid, i, j, i * numPtsAfterPadding + j, numPts, numPtsAfterPadding, numColsPadded, totalNumComparisons, index);

        if (i >= numPts || j >= numPts || j <= i)
            continue;

        cudars::Vec2d vecI = means[i];
        cudars::Vec2d vecJ = means[j];

        //            isotropicKer_.init(means[i], means[j], sigma);
        double dx, dy;
        dx = vecJ.x - vecI.x;
        dy = vecJ.y - vecI.y;
        double phi;

        //        if (dx == 0 && dy == 0) {
        //                        phi = 0.0; //mathematically undefined
        //            //            for (int k = 0; k <= numColsPadded; ++k) {
        //            //                int rowIndex = (i * numPtsAfterPadding) + j; //it's more a block index rather than row
        //            //                coeffsMat[rowIndex * numColsPadded + k] = 0.0;
        //            //            }
        ////            continue;
        //
        //        } else
        phi = atan2(dy, dx);

        double sigmaValSq = sigma1 * sigma1 + sigma2 * sigma2;
        double lambdaSqNorm = 0.25 * (dx * dx + dy * dy) / sigmaValSq;
        printf("lambdaSqNorm %f\n", lambdaSqNorm); // just to avoid seeing warning of unused variable when compiling

        //            isotropicKer_.updateFourier(arsfOrder_, coeffs_, w);
        double wNorm = 1.0 / (numPts * numPts);
        double weight = wNorm / sqrt(2.0 * M_PI * sigmaValSq);

        // updating Fourier coefficients (2 modes)
        if (pnebiMode == cudars::ArsKernelIso2dComputeMode::PNEBI_LUT)
        {
            printf("Method not fully implemented!\n");
            continue;

            //                updateARSF2CoeffRecursDownLUT(lambdaSqNorm_, phi_, w2, nFourier, pnebiLut_, coeffs);
            double cth2, sth2;
            // fastCosSin(2.0 * phi, cth2, sth2); //già commentata nell'originale
            cth2 = cos(2.0 * phi);
            sth2 = sin(2.0 * phi);

            int pnebisSz = fourierOrder + 1;
            double *pnebis = new double[pnebisSz];
            if (pnebis == nullptr)
                printf("ERROR ALLOCATING WITH NEW[]!\n");
            double sgn, cth, sth, ctmp, stmp;

            coeffsMat[0] = 0.5 * weight * pnebis[0]; // factor = w2

            sgn = -1.0;
            cth = cth2;
            sth = sth2;
            for (int k = 1; k <= fourierOrder; ++k)
            {

                coeffsMat[2 * k] = pnebis[k] * weight * sgn * cth;
                coeffsMat[2 * k + 1] = pnebis[k] * weight * sgn * sth;
                sgn = -sgn;
                ctmp = cth2 * cth - sth2 * sth;
                stmp = sth2 * cth + cth2 * sth;
                cth = ctmp;
                sth = stmp;
            }

            //            delete pnebis;
        }
        else
            printf("ERROR: pnebi mode is not LUT!\n");
    }
}

__global__ void makePartialSums(double *matIn, int nrowsIn, int ncols, double *matOut)
{
    //    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int index = threadIdx.x * gridDim.x + blockIdx.x; //!!! indexing is done "column-major" (in terms of kernel grid)
    int stride = blockDim.x * gridDim.x;

    int totalSzIn = nrowsIn * ncols; // matrix is considered of size nrows*ncols, with nrows = sumNaturalsUpToN(numPts)

    // 1 thread of the kernel composes 1 box of matOut
    int rowOutId = threadIdx.x;
    int colOutId = blockIdx.x;
    int idOut = rowOutId * ncols + colOutId;
    //    printf("rowOutId %d colOutId %d idOut %d\n", rowOutId, colOutId, idOut);

    int nOps = 0;
    for (int idx = index; idx < totalSzIn; idx += stride)
    {
        //        printf("nOps %d\n", nOps);

        matOut[idOut] += matIn[idx];
        nOps++;
    }
}

__global__ void sumColumnsPartialSums(double *matPartialSums, int nrows, int ncols, double *sums)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x; // index runs through a single block
    int stride = blockDim.x * gridDim.x;               // total number of threads in the grids

    int totalSz = nrows * ncols; // matrix is considered of size nrows*ncols, with nrows = sumNaturalsUpToN(numPts)

    for (int idx = index; idx < totalSz; idx += stride)
    {
        //        int totalIndex = (((i * nrows) + j) * ncols) + k;
        int k = idx % ncols;
        //        int rowIdx = (idx - k) / ncols; //useful for debugging
        //        printf("k %d rowIdx %d; accessing mat[%d]\n", k, rowIdx, idx);
        sums[k] += matPartialSums[idx];
    }
}

//__host__

void initParallelizationParams(ParlArsIsoParams &pp, int fourierOrder, int numPts, int blockSz, int chunkMaxSz)
{

    pp.numPts = numPts;
    int numPtsAfterPadding = numPts;
    pp.numPtsAfterPadding = numPtsAfterPadding;

    pp.chunkMaxSz = chunkMaxSz;
    int nc = numChunks(numPts, chunkMaxSz);
    pp.numChunks = nc;

    pp.blockSz = blockSz;

    const int coeffsMatNumCols = 2 * fourierOrder + 2;
    pp.coeffsMatNumCols = coeffsMatNumCols;
    const int coeffsMatNumColsPadded = coeffsMatNumCols;
    pp.coeffsMatNumColsPadded = coeffsMatNumColsPadded;
}

//__host__

void initParallelizationParams(ParlArsIsoParams &pp, int fourierOrder, int numPtsSrc, int numPtsDst, int blockSz, int chunkMaxSz)
{

    int numPts = std::max<int>(numPtsSrc, numPtsDst);
    pp.numPts = numPts;
    int numPtsAfterPadding = numPts;
    pp.numPtsAfterPadding = numPtsAfterPadding;

    pp.chunkMaxSz = chunkMaxSz;
    int nc = numChunks(numPts, chunkMaxSz);
    pp.numChunks = nc;

    pp.blockSz = blockSz;

    const int coeffsMatNumCols = 2 * fourierOrder + 2;
    pp.coeffsMatNumCols = coeffsMatNumCols;
    const int coeffsMatNumColsPadded = coeffsMatNumCols;
    pp.coeffsMatNumColsPadded = coeffsMatNumColsPadded;
}

//__host__

void updateParallelizationParams(ParlArsIsoParams &pp, int currChunkSz)
{
    // Setting up parallelization
    // Parallelization parameters
    pp.currChunkSz = currChunkSz;
    // Fourier coefficients mega-matrix computation
    const int gridTotalSize = cudars::sumNaturalsUpToN(pp.currChunkSz - 1); // total number of threads in grid Fourier coefficients grid - BEFORE PADDING
    pp.gridTotalSize = gridTotalSize;

    const int numBlocks = floor(gridTotalSize / pp.blockSz) + 1; // number of blocks in grid (each block contains blockSize threads)
    pp.numBlocks = numBlocks;
    const int gridTotalSizeAfterPadding = pp.blockSz * numBlocks;
    pp.gridTotalSizeAfterPadding = gridTotalSizeAfterPadding;

    const int coeffsMatTotalSz = gridTotalSizeAfterPadding * pp.coeffsMatNumColsPadded; // sumNaturalsUpToN(numPts - 1) * coeffsMatNumColsPadded
    pp.coeffsMatTotalSz = coeffsMatTotalSz;

    // Fourier matrix sum -> parallelization parameters
    const int sumGridSz = pp.coeffsMatNumColsPadded; // = 2 * fourierOrder + 2
    pp.sumGridSz = sumGridSz;
    const int sumBlockSz = pp.blockSz;
    pp.sumBlockSz = sumBlockSz;

    //    std::cout << "Parallelization params:" << std::endl;
    //    std::cout << "numPts " << pp.numPts << " blockSize " << pp.blockSz << " numBlocks " << numBlocks
    //            << " gridTotalSize " << gridTotalSize << " gridTotalSizeAP " << gridTotalSizeAfterPadding << std::endl;
    //    std::cout << "sumBlockSz " << sumBlockSz << " sumGridSz " << sumGridSz << std::endl;
    //        std::cout << "sum parallelization params: " << std::endl
    //            << "coeffMatNumCols " << pp.coeffsMatNumCols << " coeffsMatTotalSz " << coeffsMatTotalSz << std::endl;

    std::cout << "\nCalling kernel functions on GPU...\n"
              << std::endl;
}

//__host__

void computeArsIsoGpu(ParlArsIsoParams &paip, ArsIsoParams &arsPms, const cudars::VecVec2d &points, double *coeffsArs, hipEvent_t start, hipEvent_t stop, double &execTime)
{

    std::cout << "\n---Estimating Ars Iso---\n"
              << std::endl;

    //    double* coeffsArs = new double [paip.coeffsMatNumColsPadded];
    double *d_coeffsArs;
    hipMalloc((void **)&d_coeffsArs, paip.coeffsMatNumColsPadded * sizeof(double));
    hipMemset(d_coeffsArs, 0.0, paip.coeffsMatNumColsPadded * sizeof(double));

    for (int i = 0; i < paip.numChunks; ++i)
    {
        std::cout << "NUMPTS " << paip.numPts << std::endl;
        thrust::pair<int, int> indicesStartEnd = chunkStartEndIndices(i, paip.numPts, paip.chunkMaxSz);
        int currChunkSz = (indicesStartEnd.second - indicesStartEnd.first) + 1;

        updateParallelizationParams(paip, currChunkSz); // TODO: put chunkMaxSz as TestParams struct member?

        cudars::Vec2d *kernelInput;
        hipMalloc((void **)&kernelInput, currChunkSz * sizeof(cudars::Vec2d));
        //        hipMemcpy(kernelInput, points.data(), numPtsAfterPadding * sizeof (cudars::Vec2d), hipMemcpyHostToDevice);
        std::cout << "round " << i + 1 << "/" << paip.numChunks << " -> "
                  << "chunk-beg " << indicesStartEnd.first << " chunk-end " << indicesStartEnd.second << " --- chunk-size " << currChunkSz << std::endl;
        cudars::VecVec2d dataChunk(points.begin() + indicesStartEnd.first, points.begin() + (indicesStartEnd.first + currChunkSz));
        hipMemcpy(kernelInput, dataChunk.data(), (dataChunk.size()) * sizeof(cudars::Vec2d), hipMemcpyHostToDevice);

        // Fourier matrix sum -> parallelization parameters
        std::cout << "Parallelization params:" << std::endl;
        std::cout << "numPts " << paip.numPts << " blockSize " << paip.blockSz << " numBlocks " << paip.numBlocks
                  << " gridTotalSize " << paip.gridTotalSize << " gridTotalSizeAP " << paip.gridTotalSizeAfterPadding << std::endl;
        std::cout << "sumBlockSz " << paip.sumBlockSz << " sumGridSz " << paip.sumGridSz << std::endl;

        std::cout << "sum parallelization params: " << std::endl
                  << "coeffMatNumCols " << paip.coeffsMatNumCols << " coeffsMatTotalSz " << paip.coeffsMatTotalSz << std::endl;

        double *d_coeffsMat;
        hipMalloc((void **)&d_coeffsMat, paip.coeffsMatTotalSz * sizeof(double));
        hipMemset(d_coeffsMat, 0.0, paip.coeffsMatTotalSz * sizeof(double));

        double *d_partsums;
        hipMalloc((void **)&d_partsums, paip.blockSz * paip.coeffsMatNumColsPadded * sizeof(double));
        hipMemset(d_partsums, 0.0, paip.blockSz * paip.coeffsMatNumColsPadded * sizeof(double));

        hipEventRecord(start);
        iigDw<<<paip.numBlocks, paip.blockSz>>>(kernelInput, arsPms.arsIsoSigma, arsPms.arsIsoSigma, currChunkSz, arsPms.arsIsoOrder, paip.coeffsMatNumColsPadded, arsPms.arsIsoPnebiMode, d_coeffsMat);
        //    sumColumnsNoPadding << <1, sumBlockSz>> >(coeffsMat, gridTotalSizeAfterPadding, coeffsMatNumColsPadded, d_coeffsArs);
        makePartialSums<<<paip.coeffsMatNumColsPadded, paip.blockSz>>>(d_coeffsMat, paip.gridTotalSizeAfterPadding, paip.coeffsMatNumColsPadded, d_partsums);
        sumColumnsPartialSums<<<paip.coeffsMatNumColsPadded, 1>>>(d_partsums, paip.blockSz, paip.coeffsMatNumColsPadded, d_coeffsArs);
        hipEventRecord(stop);

        hipError_t cudaerr = hipDeviceSynchronize();
        if (cudaerr != hipSuccess)
            printf("kernel launch failed with error \"%s\".\n", hipGetErrorString(cudaerr));

        //    for (int i = 0; i < coeffsMatNumColsPadded; ++i) {
        //        std::cout << "coeffsArs[" << i << "] " << coeffsArs[i] << std::endl;
        //    }

        hipFree(d_partsums);
        hipFree(d_coeffsMat);
        hipFree(kernelInput);
    }

    hipMemcpy(coeffsArs, d_coeffsArs, paip.coeffsMatNumColsPadded * sizeof(double), hipMemcpyDeviceToHost);
    hipFree(d_coeffsArs);

    hipEventSynchronize(stop);
    float millisecondsExecTime = 0.0f;
    hipEventElapsedTime(&millisecondsExecTime, start, stop);
    std::cout << "\ninsertIsotropicGaussians() -> exec time: " << millisecondsExecTime << " ms" << std::endl;
    execTime = millisecondsExecTime;
}

void gpu_estimateRotationArsIso(const ArsImgTests::PointReaderWriter &pointsSrc, const ArsImgTests::PointReaderWriter &pointsDst, TestParams &tp, ParlArsIsoParams &paip, double &rotOut)
{
    // ARS SRC -> preparation for kernel calls and kernel calls
    hipEvent_t startSrc, stopSrc; // timing using CUDA events
    hipEventCreate(&startSrc);
    hipEventCreate(&stopSrc);

    const cudars::VecVec2d &inputSrc = pointsSrc.points();
    initParallelizationParams(paip, tp.aiPms.arsIsoOrder, inputSrc.size(), paip.blockSz, paip.chunkMaxSz); // cudarsIso.init()
    double *coeffsArsSrc = new double[paip.coeffsMatNumColsPadded];
    computeArsIsoGpu(paip, tp.aiPms, inputSrc, coeffsArsSrc, startSrc, stopSrc, paip.gpu_srcExecTime); // cudarsIso.compute()

    hipEventDestroy(startSrc);
    hipEventDestroy(stopSrc);
    // END OF ARS SRC

    //    std::cout << "\n------\n" << std::endl; //"pause" between ars src and ars dst

    // ARS DST -> preparation for kernel calls and kernel calls
    hipEvent_t startDst, stopDst; // timing using CUDA events
    hipEventCreate(&startDst);
    hipEventCreate(&stopDst);

    const cudars::VecVec2d &inputDst = pointsDst.points();
    initParallelizationParams(paip, tp.aiPms.arsIsoOrder, inputDst.size(), paip.blockSz, paip.chunkMaxSz); // cudarsIso.init()
    double *coeffsArsDst = new double[paip.coeffsMatNumColsPadded];
    computeArsIsoGpu(paip, tp.aiPms, inputDst, coeffsArsDst, startDst, stopDst, paip.gpu_dstExecTime); // cudarsIso.compute()

    hipEventDestroy(startDst);
    hipEventDestroy(stopDst);
    // END OF ARS DST

    std::cout << std::endl
              << "---Computing corelation---" << std::endl;

    // Final computations (correlation, ...) on CPU
    //     std::cout << "\nARS Coefficients:\n";
    //     std::cout << "Coefficients: Src, Dst, Cor" << std::endl;

    double thetaMax, corrMax, fourierTol;
    fourierTol = 1.0; // TODO: check for a proper tolerance

    std::vector<double> coeffsCor;
    {
        cudars::ScopedTimer("ars.correlation()");
        std::vector<double> tmpSrc;
        tmpSrc.assign(coeffsArsSrc, coeffsArsSrc + paip.coeffsMatNumColsPadded);
        std::vector<double> tmpDst;
        tmpDst.assign(coeffsArsDst, coeffsArsDst + paip.coeffsMatNumColsPadded);
        cudars::computeFourierCorr(tmpSrc, tmpDst, coeffsCor);
        cudars::findGlobalMaxBBFourier(coeffsCor, 0.0, M_PI, tp.aiPms.arsIsoThetaToll, fourierTol, thetaMax, corrMax);
        rotOut = thetaMax; //!! rotOut is passed to the function as reference
    }

    //  Output coeffs check: CPU version
    //    arsSrc.setCoefficients(coeffsArsSrc, paip.coeffsMatNumCols);
    //    //    for (int i = 0; i < coeffsVectorMaxSz; i++) {
    //    //        std::cout << "arsSrc - coeff_d[" << i << "] " << d_coeffsMat1[i] << std::endl;
    //    //    }
    //    arsDst.setCoefficients(coeffsArsDst, paip.coeffsMatNumCols);
    //    for (int i = 0; i < arsSrc.coefficients().size() && i < arsDst.coefficients().size(); ++i) {
    //        std::cout << "\t" << i << " \t" << arsSrc.coefficients().at(i) << " \t" << arsDst.coefficients().at(i) << " \t" << coeffsCor[i] << std::endl;
    //    }

    //  Output coeffs check: GPU version
    //    for (int i = 0; i < paip.coeffsMatNumCols; ++i) {
    //        std::cout << "\t" << i << " \t" << coeffsArsSrc[i] << " \t" << coeffsArsDst[i] << " \t" << coeffsCor[i] << std::endl;
    //    }
    //    std::cout << std::endl;

    std::cout << std::endl
              << "ROT OUT " << rotOut << std::endl;

    // Computes the rotated points,centroid, affine transf matrix between src and dst
    ArsImgTests::PointReaderWriter pointsRot(pointsSrc.points());
    cudars::Vec2d centroidSrc = pointsSrc.computeCentroid();
    cudars::Vec2d centroidDst = pointsDst.computeCentroid();
    cudars::Affine2d rotSrcDst = ArsImgTests::PointReaderWriter::coordToTransform(0.0, 0.0, rotOut);
    //    cudars::Vec2d translSrcDst = centroidDst - rotSrcDst * centroidSrc;
    cudars::Vec2d translSrcDst;
    cudars::vec2diff(translSrcDst, centroidDst, cudars::aff2TimesVec2WRV(rotSrcDst, centroidSrc));
    //    std::cout << "centroidSrc " << centroidSrc.x << " \t" << centroidSrc.y << "\n"
    //            << "centroidDst " << centroidDst.x << " \t" << centroidDst.y << "\n"
    //            << "rotSrcDst\n" << rotSrcDst << "\n"
    //            << "translation: [" << translSrcDst.x << " \t" << translSrcDst.y << "] rotation[deg] " << (180.0 / M_PI * rotOut) << "\n";
    pointsRot.applyTransform(translSrcDst.x, translSrcDst.y, rotOut);

    //    double rotTrue = pointsDst.getRotTheta() - pointsSrc.getRotTheta();
    //    std::cout << "\n***\npointsDst.getrotTheta() [deg]" << (180 / M_PI * pointsDst.getRotTheta())
    //            << ", pointsSrc.getrotTheta() [deg] " << (180.0 / M_PI * pointsSrc.getRotTheta()) << "\n";
    //    std::cout << "rotTrue[deg] \t" << (180.0 / M_PI * rotTrue) << " \t" << (180.0 / M_PI * cudars::mod180(rotTrue)) << std::endl;
    //    std::cout << "rotArs[deg] \t" << (180.0 / M_PI * rotOut) << " \t" << (180.0 / M_PI * cudars::mod180(rotOut)) << std::endl;

    // Free CPU memory
    delete coeffsArsSrc;
    delete coeffsArsDst;
}