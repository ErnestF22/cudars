#include "hip/hip_runtime.h"

/**
 * ARS - Angular Radon Spectrum 
 * Copyright (C) 2017 Dario Lodi Rizzini.
 *
 * ARS is free software: you can redistribute it and/or modify
 * it under the terms of the GNU Lesser General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 * 
 * ARS is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU Lesser General Public License for more details.
 *
 * You should have received a copy of the GNU Lesser General Public License
 * along with ARS.  If not, see <http://www.gnu.org/licenses/>.
 */


#include <iostream>
#include <chrono>

#include "ars/cuArsIso.cuh"



#define PRINT_DIM(X) std::cout << #X << " rows " << X.rows() << " cols " << X.cols() << std::endl;
#define RAD2DEG(X) (180.0/M_PI*(X))

struct BoundInterval {
    double x0;
    double x1;
    double y0;
    double y1;
};

void rangeToPoint(double* ranges, int num, int numPadded, double angleMin, double angleRes, std::vector<cuars::Vec2d>& points);

int main(void) {
    double acesRanges[] = {50.00, 50.00, 50.00, 5.26, 5.21, 5.06, 5.01, 3.01, 2.94, 2.89, 2.84, 2.74, 2.69, 2.64, 2.59, 2.54, 2.49, 2.49, 2.44, 2.39, 2.34, 2.29, 2.29, 2.29, 2.39, 2.39, 2.49, 2.51, 2.61, 2.66, 2.76, 2.81, 2.96, 3.01, 3.11, 3.26, 3.01, 3.01, 3.01, 3.06, 3.21, 6.86, 6.86, 6.81, 6.76, 6.71, 6.71, 6.66, 6.61, 6.66, 6.56, 6.56, 6.56, 6.46, 6.46, 6.41, 6.46, 6.46, 4.11, 3.96, 3.96, 4.96, 4.86, 5.21, 7.41, 4.61, 5.16, 6.26, 6.26, 6.31, 4.86, 5.01, 5.86, 5.81, 4.21, 4.26, 4.31, 4.41, 4.39, 4.46, 5.31, 5.06, 5.26, 4.96, 6.01, 5.76, 5.61, 5.36, 5.26, 5.01, 4.21, 4.16, 4.01, 3.91, 3.61, 3.21, 3.26, 3.16, 3.06, 3.01, 3.31, 3.21, 3.16, 2.16, 2.19, 2.16, 2.21, 2.11, 2.01, 2.01, 2.06, 2.84, 2.91, 2.91, 3.01, 3.11, 3.21, 3.81, 4.06, 7.11, 7.06, 7.01, 6.96, 6.86, 4.31, 6.76, 6.71, 6.66, 6.61, 5.46, 5.41, 6.46, 6.21, 6.31, 6.51, 7.26, 7.46, 50.00, 2.01, 1.94, 1.94, 1.94, 2.31, 1.86, 1.84, 1.84, 1.81, 1.96, 26.46, 20.76, 2.11, 2.12, 2.17, 2.14, 2.09, 2.09, 2.14, 2.14, 2.14, 2.14, 2.14, 2.14, 2.14, 2.14, 2.14, 2.19, 2.19, 2.24, 2.24, 2.24, 2.24, 2.29, 2.29, 2.29, 2.29, 2.29, 2.39, 2.39, 2.39, 2.44};
    cuars::AngularRadonSpectrum2d ars1;
    cuars::AngularRadonSpectrum2d ars2;
    //    std::chrono::system_clock::time_point timeStart, timeStop;
    double sigma = 0.05;
    int fourierOrder = 20;

    ars1.setARSFOrder(fourierOrder);
    ars2.setARSFOrder(fourierOrder);

    //parallelization parameters
    int numPts = 180; // = acesRanges.size()
    const int numPtsAfterPadding = ceilPow2(numPts);
    const int blockSize = 256; //num threads per block
    const int numBlocks = (numPtsAfterPadding * numPtsAfterPadding) / blockSize; //number of blocks in grid (each block contains blockSize threads)
    const int gridTotalSize = blockSize*numBlocks; //total number of threads in grid

    const int sumBlockSz = 64;
    const int sumGridSz = 256;
    std::cout << "Parallelization params:" << std::endl;
    std::cout << "numPtsAfterPadding " << numPtsAfterPadding << " blockSize " << blockSize << " numBlocks " << numBlocks << " gridTotalSize " << gridTotalSize << std::endl;
    std::cout << "sumSrcBlockSz " << sumBlockSz << " sumGridSz " << sumGridSz << std::endl;
    std::cout << "numPtsAfterPadding " << numPtsAfterPadding << " blockSize " << blockSize << " numBlocks " << numBlocks << " gridTotalSize " << gridTotalSize << std::endl;

    //conversion
    std::vector<cuars::Vec2d> acesPointsSTL;
    //    cuars::Vec2d p0, p1;
    //    p0.x = 0.0;
    //    p0.y = 0.0;
    //    p1.x = cos(M_PI * 30 / 180.0);
    //    p1.y = sin(M_PI * 30 / 180.0);
    //    acesPointsSTL.push_back(p0);
    //    acesPointsSTL.push_back(p1);

    rangeToPoint(acesRanges, numPts, numPtsAfterPadding, -0.5 * M_PI, M_PI / 180.0 * 1.0, acesPointsSTL);

    thrust::host_vector<cuars::Vec2d> acesPointsHost(acesPointsSTL.begin(), acesPointsSTL.end());


    //    cuars::Vec2d firstElement; //??

    std::cout << "\n------\n" << std::endl;
    std::cout << "\n\nCalling kernel functions on GPU\n" << std::endl;

    //    timeStart = std::chrono::system_clock::now();
    //ars1 kernel call
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    //    ars1.insertIsotropicGaussians(acesPoints1, sigma);
    cuars::Vec2d * kernelInput1;
    hipMalloc((void**) &kernelInput1, numPtsAfterPadding * sizeof (cuars::Vec2d));
    hipMemcpy(kernelInput1, acesPointsHost.data(), numPtsAfterPadding * sizeof (cuars::Vec2d), hipMemcpyHostToDevice);
    //    for (int i = 0; i < numPtsAfterPadding; ++i) {
    //        kernelInput1[i] = acesPointsSTL[i];
    //    }

    //    hipDeviceSynchronize();
    //    std::cout << "acesPointsHost.size() " << acesPointsHost.size() << std::endl;
    //    for (int s = 0; s < acesPointsHost.size(); s++) {
    //        std::cout << "s " << s << std::endl;
    //        std::cout << kernelInput1[s].x << " " << kernelInput1[s].y << std::endl;
    //    }


    //    ars1.initLUT(0.0001);
    cuars::ArsKernelIsotropic2d::ComputeMode pnebiMode = cuars::ArsKernelIsotropic2d::ComputeMode::PNEBI_DOWNWARD;
    ars1.setComputeMode(pnebiMode);


    const int coeffsMatNumCols = 2 * fourierOrder + 2;
    const int coeffsMatNumColsPadded = ceilPow2(coeffsMatNumCols);
    const int coeffsMatTotalSz = numPtsAfterPadding * numPtsAfterPadding * coeffsMatNumColsPadded;
    double *coeffsMat1;
    hipMalloc((void**) &coeffsMat1, coeffsMatTotalSz * sizeof (double));
    hipMemset(coeffsMat1, 0.0, coeffsMatTotalSz * sizeof (double));
    //    for (int i = 0; i < coeffsMatTotalSz; ++i) {
    //        coeffsMat1[i] = 0.0;
    //    }
    double* d_coeffsArs1;
    hipMalloc((void**) &d_coeffsArs1, coeffsMatNumColsPadded * sizeof (double));
    hipMemset(d_coeffsArs1, 0.0, coeffsMatNumColsPadded * sizeof (double));


    //    cuars::PnebiLUT pnebiLUT1; //LUT setup
    //    double lutPrecision = 0.001; //LUT setup
    //    pnebiLUT1.init(fourierOrder, lutPrecision); //LUT setup
    //    if (pnebiLUT1.getOrderMax() < fourierOrder) { //LUT setup
    //        ARS_ERROR("LUT not initialized to right order. Initialized now."); //LUT setup
    //        pnebiLUT1.init(fourierOrder, 0.0001); //LUT setup
    //    }

    hipEventRecord(start);
    iigKernelDownward_old << <numBlocks, blockSize >> >(kernelInput1, sigma, sigma, numPts, numPtsAfterPadding, fourierOrder, coeffsMatNumColsPadded, pnebiMode, coeffsMat1);
    sumColumns << <1, sumBlockSz>> >(coeffsMat1, numPtsAfterPadding, coeffsMatNumColsPadded, d_coeffsArs1);
    hipEventRecord(stop);

    double* coeffsArs1 = new double [coeffsMatNumColsPadded];
    hipMemcpy(coeffsArs1, d_coeffsArs1, coeffsMatNumColsPadded * sizeof (double), hipMemcpyDeviceToHost);

    hipEventSynchronize(stop);
    float millisecondsArs1 = 0.0;
    hipEventElapsedTime(&millisecondsArs1, start, stop);

    hipEventDestroy(start);
    hipEventDestroy(stop);
    //END OF ARS1

    std::cout << "\n------\n" << std::endl;


    //ARS2    
    ars2.setComputeMode(cuars::ArsKernelIsotropic2d::ComputeMode::PNEBI_LUT);

    cuars::PnebiLUT pnebiLUT2; //LUT setup
    double lutPrecision = 0.001; //already initialized for pnebiLUT1
    pnebiLUT2.init(fourierOrder, lutPrecision); //LUT setup
    if (pnebiLUT2.getOrderMax() < fourierOrder) { //LUT setup
        ARS_ERROR("LUT not initialized to right order. Initialized now."); //LUT setup
        pnebiLUT2.init(fourierOrder, 0.0001); //LUT setup
    }


    //kernel call
    //    timeStart = std::chrono::system_clock::now();
    hipEventCreate(&start);
    hipEventCreate(&stop);

    //    ars2.insertIsotropicGaussians(acesPoints1, sigma);

    cuars::Vec2d* kernelInput2;
    hipMalloc((void **) &kernelInput2, numPtsAfterPadding * sizeof (cuars::Vec2d));
    pnebiMode = cuars::ArsKernelIsotropic2d::ComputeMode::PNEBI_LUT;

    double *coefficientsArs2 = new double[coeffsMatTotalSz](); //() initialize to 0
    double *d_coefficientsArs2; //d_ stands for device
    //    const int coeffsVectorMaxSz = 2 * fourierOrder + 2; //already initialized in ars1
    hipMalloc(&d_coefficientsArs2, coeffsMatTotalSz * sizeof (double)); //maybe directly use hipMemset?
    hipMemcpy(d_coefficientsArs2, coefficientsArs2, coeffsMatTotalSz * sizeof (double), hipMemcpyHostToDevice);


    hipEventRecord(start);
    //    iigKernel << < numBlocks, blockSize >> >(thrust::raw_pointer_cast<ars::Vec2d*>(kernelInput2.data()), sigma, sigma, numPts, paddedPtVecSz, fourierOrder, pnebiMode, pnebiLUT2, d_coefficientsArs2);
    //    sumColumns << <1, sumBlockSz>> >(coeffsMat2, numPtsAfterPadding, coeffsMatNumColsPadded, d_coeffsArs2);
    hipEventRecord(stop);
    //end of kernel calls for ARS2

    hipMemcpy(coefficientsArs2, d_coefficientsArs2, coeffsMatTotalSz * sizeof (double), hipMemcpyDeviceToHost);

    hipEventSynchronize(stop);
    float millisecondsArs2 = 0.0;
    hipEventElapsedTime(&millisecondsArs2, start, stop);

    //    timeStop = std::chrono::system_clock::now();
    //    double timeArs2 = (double) std::chrono::duration_cast<std::chrono::milliseconds>(timeStop - timeStart).count();
    //    std::cout << "insertIsotropicGaussians() " << timeArs2 << " ms" << std::endl;
    hipEventDestroy(start);
    hipEventDestroy(stop);
    //END OF ARS2


    hipError_t cudaerr = hipDeviceSynchronize();
    if (cudaerr != hipSuccess)
        printf("kernel launch failed with error \"%s\".\n",
            hipGetErrorString(cudaerr));

    std::cout << "ARS1 execution " << millisecondsArs1 << " ms --- ARS2 execution " << millisecondsArs2 << " ms" << std::endl;



    std::cout << "\nARS Coefficients:\n";
    std::cout << "\ti \tDownward \tLUT\n";
    ars1.setCoefficients(coeffsArs1, coeffsMatNumCols);
    //    for (int i = 0; i < coeffsVectorMaxSz; i++) {
    //        std::cout << "ars1coeff_d[" << i << "] " << d_coeffsMat1[i] << std::endl;
    //    }
    ars2.setCoefficients(coefficientsArs2, coeffsMatNumCols);
    for (int i = 0; i < ars1.coefficients().size() && i < ars2.coefficients().size(); ++i) {
        std::cout << "\t" << i << " \t" << ars1.coefficients().at(i) << " \t" << ars2.coefficients().at(i) << "\n";
    }
    std::cout << std::endl;

    std::vector<double> funcFourierRecursDownLUT;
    std::vector<double> funcFourierRecursDown;
    int thnum = 360;
    double dtheta = M_PI / thnum;
    double theta;
    for (int i = 0; i < thnum; ++i) {
        theta = dtheta * i;
        funcFourierRecursDownLUT.push_back(ars1.eval(theta));
        funcFourierRecursDown.push_back(ars2.eval(theta));
    }

    std::cout << "\nBranch and Bound limits:\n";
    int bbnum = 32;
    std::vector<BoundInterval> bbbs(bbnum);
    for (int i = 0; i < bbnum; ++i) {
        bbbs[i].x0 = M_PI * i / bbnum;
        bbbs[i].x1 = M_PI * (i + 1) / bbnum;
        cuars::findLUFourier(ars1.coefficients(), bbbs[i].x0, bbbs[i].x1, bbbs[i].y0, bbbs[i].y1);
        std::cout << i << ": x0 " << RAD2DEG(bbbs[i].x0) << " x1 " << RAD2DEG(bbbs[i].x1) << ", y0 " << bbbs[i].y0 << " y1 " << bbbs[i].y1 << std::endl;
    }


    cuars::FourierOptimizerBB1D optim(ars1.coefficients());
    double xopt, ymin, ymax;
    optim.enableXTolerance(true);
    optim.enableYTolerance(true);
    optim.setXTolerance(M_PI / 180.0 * 0.5);
    optim.setYTolerance(1.0);
    optim.findGlobalMax(0, M_PI, xopt, ymin, ymax);
    std::cout << "\n****\nMaximum in x = " << xopt << " (" << RAD2DEG(xopt) << " deg), maximum between [" << ymin << "," << ymax << "]" << std::endl;

    double xopt2, ymax2;
    cuars::findGlobalMaxBBFourier(ars1.coefficients(), 0, M_PI, M_PI / 180.0 * 0.5, 1.0, xopt2, ymax2);
    std::cout << "  repeated evaluation with findGlobalMaxBBFourier(): maximum in x " << xopt2 << " (" << RAD2DEG(xopt2) << " deg), maximum value " << ymax2 << std::endl;



    //    //Free GPU and CPU memory
    hipFree(d_coefficientsArs2);
    hipFree(kernelInput2);
    delete coefficientsArs2;
    //    free(coefficientsArs2); //cpu array
    hipFree(coeffsMat1);
    hipFree(kernelInput1);
    hipFree(d_coeffsArs1);
    delete coeffsArs1;

    return 0;
}

void rangeToPoint(double* ranges, int num, int numPadded, double angleMin, double angleRes, std::vector<cuars::Vec2d>& points) {
    cuars::Vec2d p;
    for (int i = 0; i < numPadded; ++i) {
        if (i < num) {
            double a = angleMin + angleRes * i;
            p.x = ranges[i] * cos(a);
            p.y = ranges[i] * sin(a);
            points.push_back(p);
        } else {
            //padding with zeros

            p.x = 0.0;
            p.y = 0.0;
            points.push_back(p);
        }
        //        std::cout << p.x << " " << p.y << std::endl;
    }
}


