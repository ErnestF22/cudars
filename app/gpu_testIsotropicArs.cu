#include "hip/hip_runtime.h"

/**
 * ARS - Angular Radon Spectrum 
 * Copyright (C) 2017 Dario Lodi Rizzini.
 *
 * ARS is free software: you can redistribute it and/or modify
 * it under the terms of the GNU Lesser General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 * 
 * ARS is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU Lesser General Public License for more details.
 *
 * You should have received a copy of the GNU Lesser General Public License
 * along with ARS.  If not, see <http://www.gnu.org/licenses/>.
 */
#include <iostream>

#include <ars/definitions.h>
#include <ars/ars2d.h>
#include <ars/BBOptimizer1d.h>

#include <thrust/host_vector.h>
#include <thrust/device_malloc.h>

#include <chrono>

#include <>



#define PRINT_DIM(X) std::cout << #X << " rows " << X.rows() << " cols " << X.cols() << std::endl;
#define RAD2DEG(X) (180.0/M_PI*(X))

struct BoundInterval {
    double x0;
    double x1;
    double y0;
    double y1;
};

void rangeToPoint(double* ranges, int num, int numPadded, double angleMin, double angleRes, std::vector<cuars::Vec2d>& points);

int ceilPow2(int n);

//__device__

double evaluatePnebi0Polynom(double x) {
    double t, t2, tinv, val;

    if (x < 0.0) x = -x;
    t = x / 3.75;

    if (t < 1.0) {
        t2 = t*t;
        val = 1.0 + t2 * (3.5156229 + t2 * (3.0899424 + t2 * (1.2067492 + t2 * (0.2659732 + t2 * (0.360768e-1 + t2 * 0.45813e-2)))));
        val = 2.0 * exp(-x) * val;
    } else {
        tinv = 1 / t;
        val = (0.39894228 + tinv * (0.1328592e-1 + tinv * (0.225319e-2 + tinv * (-0.157565e-2 + tinv *
                (0.916281e-2 + tinv * (-0.2057706e-1 + tinv * (0.2635537e-1 + tinv * (-0.1647633e-1 + tinv * 0.392377e-2))))))));
        val = 2.0 * val / sqrt(x);
    }

    return val;
}

//__device__

void evaluatePnebiVectorGPU(int n, double x, double* pnebis, int pnebisSz) {
    double factor, seqPrev, seqCurr, seqNext;
    //    if (pnebis.size() < n + 1) { //questa condizione dovrebbe essere già garantita prima della chiamata di evaluatePnebiVectorGPU
    //        pnebis.resize(n + 1); //ovvero: il questo resizing non dovrebbe essere necessario
    //    }

    if (x < 0.0) x = -x;

    // If x~=0, then BesselI(0,x) = 1.0 and BesselI(k,x) = 0.0 for k > 0.
    // Thus, PNEBI(0,x) = 2.0 and PNEBI(k,x) = 0.0 for k > 0.
    //TODO 9): this if should be done in iigKernel before calling present function
    if (x < 1e-6) {
        pnebis[0] = 2.0;
        for (int i = 1; i < pnebisSz; ++i)
            pnebis[i] = 0.0;
        return;
    }

    // Computes bessel function using back recursion
    factor = 2.0 / x;
    seqPrev = 0.0; // bip
    seqCurr = 1.0; // bi
    seqNext = 0.0; // bim
    for (int k = 2 * (n + (int) sqrt(40.0 * n)); k >= 0; --k) {
        seqNext = seqPrev + factor * k * seqCurr;
        seqPrev = seqCurr;
        seqCurr = seqNext;
        if (k <= n) {
            pnebis[k] = seqPrev;
        }
        // To avoid overflow!
        if (seqCurr > cuars::BIG_NUM) {
            seqPrev *= cuars::SMALL_NUM;
            seqCurr *= cuars::SMALL_NUM;
            for (int i = 0; i < pnebisSz; ++i) {
                pnebis[i] *= cuars::SMALL_NUM;
            }
            //std::cerr << __FILE__ << "," << __LINE__ << ": ANTI-OVERFLOW!" << std::endl;
        }
    }

    double scaleFactor = evaluatePnebi0Polynom(x) / pnebis[0];
    for (int i = 0; i < pnebisSz; ++i) {
        pnebis[i] = scaleFactor * pnebis[i];
    }
}

//__global__

void iigKernel(cuars::Vec2d* means, double sigma1, double sigma2, int numPts, int numPtsAfterPadding, int fourierOrder, int numColsPadded, cuars::ArsKernelIsotropic2d::ComputeMode pnebiMode, cuars::PnebiLUT& pnebiLUT, double* coeffsMat) {
    //    a.insertIsotropicGaussians(points, sigma);

    //TODO 1): 4 righe sotto: DA FARE NEL MAIN PRIMA DI CHIAMARE LA FUNZIONE; vengono fatte una tantum prima del for
    //    if (coeffs_.size() != 2 * arsfOrder_ + 2) {
    //        coeffs_.resize(2 * arsfOrder_ + 2);
    //    }
    //    std::fill(coeffs_.begin(), coeffs_.end(), 0.0);

    //    int index = blockIdx.x * blockDim.x + threadIdx.x; //index runs through a single block
    //    int stride = blockDim.x * gridDim.x; //total number of threads in the grid

    const int totalNumComparisons = numPtsAfterPadding * numPtsAfterPadding;

    for (int tid = 0; tid < totalNumComparisons; ++tid) {

        int j = tid % numPtsAfterPadding;
        int i = (tid - j) / numPtsAfterPadding;
        printf("i %d j %d\n", i, j);
        //        printf("tid %d i %d j %d tidIJ %d --- numPts %d numPtsAfterPadding %d numColsPadded %d totNumComp %d\n", tid, i, j, i * numPtsAfterPadding + j, numPts, numPtsAfterPadding, numColsPadded, totalNumComparisons);

        if (i >= numPts || j >= numPts)
            continue;

        cuars::Vec2d vecI = means[i];
        cuars::Vec2d vecJ = means[j];

        //            isotropicKer_.init(means[i], means[j], sigma);
        double dx, dy;
        dx = vecJ.x - vecI.x;
        dy = vecJ.y - vecI.y;
        double phi;

        if (dx == 0 && dy == 0) {
            //            phi = 0.0; //mathematically undefined
            //            for (int k = 0; k <= numColsPadded; ++k) {
            //                int rowIndex = (i * numPtsAfterPadding) + j; //it's more a block index rather than row 
            //                coeffsMat[rowIndex * numColsPadded + k] = 0.0;
            //            }
            continue;

        } else
            phi = atan2(dy, dx);

        double sigmaValSq = sigma1 * sigma1 + sigma2 * sigma2;
        double lambdaSqNorm = 0.25 * (dx * dx + dy * dy) / sigmaValSq;


        //            isotropicKer_.updateFourier(arsfOrder_, coeffs_, w);
        double weight = 1.0 / (numPts * numPts);
        double w2 = weight / sqrt(2.0 * M_PI * sigmaValSq);

        //TODO 2): TROVARE UNA SOLUZIONE A QUESTO RESIZING (farlo prima di dimensione fissa sufficiente nel main?)
        //            if (coeffs.size() != 2 * nFourier + 2) {
        //                coeffs.resize(2 * nFourier + 2);
        //            }

        //TODO 3): fare questa inizializzazione della LUT nel main
        //            if (pnebiLut_.getOrderMax() < nFourier) {
        //                ARS_ERROR("LUT not initialized to right order. Initialized now.");
        //                pnebiLut_.init(nFourier, 0.0001);
        //            }

        //updating Fourier coefficients (2 modes)
        if (pnebiMode == cuars::ArsKernelIsotropic2d::ComputeMode::PNEBI_DOWNWARD) {
            //                updateARSF2CoeffRecursDown(lambdaSqNorm, phi, w2, nFourier, coeffs);

            double cth2, sth2;
            cth2 = cos(2.0 * phi);
            sth2 = sin(2.0 * phi);
            //                updateARSF2CoeffRecursDown(lambda, cth2, sth2, factor, n, coeffs);


            //TODO 4): make pnebis a double*
            //can solve it with cuda/gpu malloc here? otherwise just pass the needed pointer to the function?
            //for now I just declare it here
            //                std::vector<double> pnebis(n + 1);
            int pnebisSz = fourierOrder + 1;
            double *pnebis = new double[pnebisSz];

            double sgn, cth, sth, ctmp, stmp;

            // Fourier Coefficients 
            //                if (coeffs.size() != 2 * n + 2) {
            //                    std::cerr << __FILE__ << "," << __LINE__ << ": invalid size of Fourier coefficients vector " << coeffs.size() << " should be " << (2 * n + 2) << std::endl;
            //                    return;
            //                }

            //                                TODO 5): expand evaluatePnebiVector() below
            evaluatePnebiVectorGPU(fourierOrder, lambdaSqNorm, pnebis, pnebisSz);
            //                ARS_PRINT(pnebis[0]);

            //!!!! factor = w2
            double factor = w2;
            int rowIndex = (i * numPtsAfterPadding) + j; // = tid
            coeffsMat[rowIndex * numColsPadded + 0] += 0.5 * factor * pnebis[0];
            std::cout << "coeff0" << 0.5 * factor * pnebis[0] << std::endl;


            sgn = -1.0;
            cth = cth2;
            sth = sth2;
            //!!!! n in the for below is fourierOrder
            //                for (int k = 1; k <= n; ++k) {
            for (int k = 1; k <= fourierOrder; ++k) {
                std::cout << "coeff" << 2 * k << " " << factor * pnebis[k] * sgn * cth << std::endl;
                std::cout << "coeff" << 2 * k + 1 << " " << factor * pnebis[k] * sgn * cth << std::endl;
                coeffsMat[(rowIndex * numColsPadded) + (2 * k)] += factor * pnebis[k] * sgn * cth;
                coeffsMat[(rowIndex * numColsPadded) + ((2 * k) + 1)] += factor * pnebis[k] * sgn * sth;
                sgn = -sgn;
                ctmp = cth2 * cth - sth2 * sth;
                stmp = sth2 * cth + cth2 * sth;
                cth = ctmp;
                sth = stmp;
            }
        } else if (pnebiMode == cuars::ArsKernelIsotropic2d::ComputeMode::PNEBI_LUT) {
            //                updateARSF2CoeffRecursDownLUT(lambdaSqNorm_, phi_, w2, nFourier, pnebiLut_, coeffs);
            double cth2, sth2;
            //fastCosSin(2.0 * phi, cth2, sth2); //già commentata nell'originale
            cth2 = cos(2.0 * phi);
            sth2 = sin(2.0 * phi);

            //TODO 6): find a workaround for this pnebis vector; for now I just initialize here a double* pnebis;
            //                std::vector<double> pnebis(fourierOrder + 1); //prima riga della funzione omonima chiamata da dentro l'inline
            int pnebisSz = fourierOrder + 1;
            double *pnebis = new double[pnebisSz];
            double sgn, cth, sth, ctmp, stmp;

            //TODO 7): minor problem... seems just to be a check of standing conditions. Still... might be useful to understand it in order to fix dimensions of pointers passed to iigKernel
            // Fourier Coefficients 
            //                if (coeffs.size() != 2 * fourierOrder + 2 || pnebiLUT.getOrderMax() < fourierOrder) {
            //                    std::cerr << __FILE__ << "," << __LINE__ << ": one of these conditions failed:"
            //                            << "\n  size of Fourier coefficients vector " << coeffs.size() << " should be " << (2 * n + 2)
            //                            << "\n  LUT max order is " << pnebiLUT.getOrderMax() << " >= " << n
            //                            << std::endl;
            //                    return;
            //                }

            // TODO 8): SOLVE PROBLEM OF FUNCTION COMMENTED BELOW (NOTE THAT ITS CODE HAS ALREADY BEEN COPIED IN THE SCOPE BELOW THE COMMENTED CALLING OF THE FUNCTION)
            //                pnebiLUT.eval(lambdaSqNorm, pnebis);
            //                evalPnebiLUT2();
            //                //ARS_PRINT(pnebis[0]);


            coeffsMat[0] = 0.5 * w2 * pnebis[0]; //factor = w2

            sgn = -1.0;
            cth = cth2;
            sth = sth2;
            for (int k = 1; k <= fourierOrder; ++k) {

                coeffsMat[2 * k] = pnebis[k] * w2 * sgn * cth;
                coeffsMat[2 * k + 1] = pnebis[k] * w2 * sgn * sth;
                sgn = -sgn;
                ctmp = cth2 * cth - sth2 * sth;
                stmp = sth2 * cth + cth2 * sth;
                cth = ctmp;
                sth = stmp;
            }
        }



    }
}

int main(void) {
    double acesRanges[] =  {4.32, 1.13, 3.51, 2.54, 4.25, 2.17, 4.85, 1.27, 7.24, 9.43, 1.36, 6.30, 8.36, 7.61, 0.31, 7.49, 0.38, 7.23, 3.97, 0.54, 1.38, 9.44, 5.93, 7.57, 3.96, 8.19, 1.44, 1.73, 8.01, 3.85, 4.58, 3.71, 2.28, 3.79, 5.43, 3.57, 9.24, 8.47, 4.52, 0.60, 1.07, 0.56, 3.26, 8.90, 0.95, 4.48, 8.78, 1.41, 4.63, 0.64, 4.19, 6.31, 9.86, 1.68, 9.03, 6.51, 8.70, 7.58, 0.10, 1.35, 3.06, 1.72, 5.98, 3.66, 1.18, 5.54, 8.98, 3.52, 6.17, 7.10, 6.26, 4.23, 6.18, 2.06, 4.27, 2.21, 7.52, 6.30, 8.71, 3.17, 8.56, 4.65, 0.16, 2.02, 7.05, 6.34, 6.37, 0.66, 4.33, 1.10, 9.50, 4.68, 4.72, 4.55, 0.69, 7.38, 3.77, 3.22, 1.43, 9.43, 6.41, 5.53, 7.00, 4.61, 5.42, 3.80, 1.73, 5.78, 0.45, 6.42, 4.99, 9.01, 8.86, 1.86, 4.33, 4.29, 3.00, 3.60, 4.66, 7.72, 5.54, 0.75, 2.48, 2.14, 7.02, 2.14, 1.10, 4.36, 4.06, 6.81, 1.24, 7.58, 9.29, 9.41, 0.83, 9.31, 0.24, 1.57, 6.17, 9.61, 3.38, 9.74, 6.89, 2.30, 5.51, 9.17, 5.11, 8.25, 0.72, 0.17, 8.61, 7.46, 6.39, 1.24, 7.01, 3.94, 0.08, 4.82, 3.86, 1.05, 3.05, 5.37, 6.21, 2.88, 6.86, 6.00, 1.17, 8.85, 0.23, 2.72, 7.51, 8.84, 6.77, 7.18, 1.79, 8.43, 3.02, 5.86, 5.36, 9.83};
    cuars::AngularRadonSpectrum2d ars1;
    cuars::AngularRadonSpectrum2d ars2;
    std::chrono::system_clock::time_point timeStart, timeStop;
    double sigma = 0.05;
    int fourierOrder = 20;

    ars1.setARSFOrder(fourierOrder);
    ars2.setARSFOrder(fourierOrder);

    //parallelization parameters
    int numPts = 180; // = acesRanges.size()
    const int numPtsAfterPadding = ceilPow2(numPts);
    const int blockSize = 256; //num threads per block
    const int numBlocks = (numPtsAfterPadding * numPtsAfterPadding) / blockSize; //number of blocks in grid (each block contains blockSize threads)
    const int gridTotalSize = blockSize*numBlocks; //total number of threads in grid

    //conversion
    std::vector<cuars::Vec2d> acesPointsSTL;
    rangeToPoint(acesRanges, numPts, numPtsAfterPadding, -0.5 * M_PI, M_PI / 180.0 * 1.0, acesPointsSTL);

    thrust::host_vector<cuars::Vec2d> acesPointsHost(acesPointsSTL.begin(), acesPointsSTL.end());


    //    cuars::Vec2d firstElement; //??



    timeStart = std::chrono::system_clock::now();
    //ars1 kernel call
    //    ars1.insertIsotropicGaussians(acesPoints1, sigma);
    cuars::Vec2d * kernelInput1 = new cuars::Vec2d [numPtsAfterPadding];
    //    hipMallocManaged((void**) &kernelInput1, numPtsAfterPadding * sizeof (cuars::Vec2d));
    //    hipMemcpy(kernelInput1, acesPointsHost.data(), numPtsAfterPadding * sizeof (cuars::Vec2d), hipMemcpyDefault);
    for (int i = 0; i < numPtsAfterPadding; ++i) {
        kernelInput1[i] = acesPointsSTL[i];
    }

    //    hipDeviceSynchronize();
    //    std::cout << "acesPointsHost.size() " << acesPointsHost.size() << std::endl;
    //    for (int s = 0; s < acesPointsHost.size(); s++) {
    //        std::cout << "s " << s << std::endl;
    //        std::cout << kernelInput1[s].x << " " << kernelInput1[s].y << std::endl;
    //    }


    //    ars1.initLUT(0.0001);
    cuars::ArsKernelIsotropic2d::ComputeMode pnebiMode = cuars::ArsKernelIsotropic2d::ComputeMode::PNEBI_DOWNWARD;
    ars1.setComputeMode(pnebiMode);



    const int coeffsMatNumCols = 2 * fourierOrder + 2;
    const int coeffsMatNumColsPadded = ceilPow2(coeffsMatNumCols);
    const int coeffsMatTotalSz = numPtsAfterPadding * numPtsAfterPadding * coeffsMatNumColsPadded;
    double *coeffsMat1 = new double [coeffsMatTotalSz];
    //    hipMallocManaged((void**) &coeffsMat1, coeffsMatTotalSz * sizeof (double));
    //    hipMemset(coeffsMat1, 0.0, coeffsMatTotalSz * sizeof (double));
    for (int i = 0; i < coeffsMatTotalSz; ++i) {
        coeffsMat1[i] = 0.0;
    }


    cuars::PnebiLUT pnebiLUT1; //LUT setup
    double lutPrecision = 0.001; //LUT setup
    pnebiLUT1.init(fourierOrder, lutPrecision); //LUT setup
    if (pnebiLUT1.getOrderMax() < fourierOrder) { //LUT setup
        ARS_ERROR("LUT not initialized to right order. Initialized now."); //LUT setup
        pnebiLUT1.init(fourierOrder, 0.0001); //LUT setup
    }


    //    iigKernel << < 1, 1 >> >(kernelInput1, sigma, sigma, numPts, numPtsAfterPadding, fourierOrder, coeffsMatNumColsPadded, pnebiMode, pnebiLUT1, coeffsMat1);
    iigKernel(kernelInput1, sigma, sigma, numPts, numPtsAfterPadding, fourierOrder, coeffsMatNumColsPadded, pnebiMode, pnebiLUT1, coeffsMat1);

    //    hipMemcpy(coeffsMat1, d_coeffsMat1, coeffsVectorMaxSz * sizeof (double), hipMemcpyDefault);
    //end of kernel call


    double* coeffsArs1 = new double [coeffsMatNumColsPadded];
    for (int k = 0; k < coeffsMatNumColsPadded; ++k)
        coeffsArs1[k] = 0.0; //init coeffsArs vector to 0    
    for (int i = 0; i < numPtsAfterPadding; ++i)
        for (int j = 0; j < numPtsAfterPadding; ++j)
            for (int k = 0; k < coeffsMatNumColsPadded; ++k) {
                int totalIndex = (((i * numPtsAfterPadding) + j) * coeffsMatNumColsPadded) + k;
                coeffsArs1[k] += coeffsMat1[totalIndex];
            }
    for (int i = 0; i < coeffsMatNumColsPadded; ++i) {
        std::cout << "coeffsArs1[" << i << "] " << coeffsArs1[i] << std::endl;
    }

    timeStop = std::chrono::system_clock::now();
    double timeArs1 = (double) std::chrono::duration_cast<std::chrono::milliseconds>(timeStop - timeStart).count();
    hipDeviceSynchronize();
    std::cout << "insertIsotropicGaussians() " << timeArs1 << " ms" << std::endl;
    //END OF ARS1

    std::cout << "\n------\n" << std::endl;


    //ARS2    
    ars2.setComputeMode(cuars::ArsKernelIsotropic2d::ComputeMode::PNEBI_LUT);

    timeStart = std::chrono::system_clock::now();

    //kernel call
    //    ars2.insertIsotropicGaussians(acesPoints1, sigma);
    cuars::Vec2d* kernelInput2;
    hipMalloc((void **) &kernelInput2, numPtsAfterPadding * sizeof (cuars::Vec2d));
    pnebiMode = cuars::ArsKernelIsotropic2d::ComputeMode::PNEBI_LUT;

    double *coefficientsArs2 = new double[coeffsMatTotalSz](); //() initialize to 0
    double *d_coefficientsArs2; //d_ stands for device
    //    const int coeffsVectorMaxSz = 2 * fourierOrder + 2; //already initialized in ars1
    hipMalloc(&d_coefficientsArs2, coeffsMatTotalSz * sizeof (double)); //maybe directly use hipMemset?
    hipMemcpy(d_coefficientsArs2, coefficientsArs2, coeffsMatTotalSz * sizeof (double), hipMemcpyHostToDevice);

    cuars::PnebiLUT pnebiLUT2; //LUT setup
    //    double lutPrecision = 0.001; //already initialized for pnebiLUT1
    pnebiLUT2.init(fourierOrder, lutPrecision); //LUT setup
    if (pnebiLUT2.getOrderMax() < fourierOrder) { //LUT setup
        ARS_ERROR("LUT not initialized to right order. Initialized now."); //LUT setup
        pnebiLUT2.init(fourierOrder, 0.0001); //LUT setup
    }

    //    iigKernel << < numBlocks, blockSize >> >(thrust::raw_pointer_cast<ars::Vec2d*>(kernelInput2.data()), sigma, sigma, numPts, paddedPtVecSz, fourierOrder, pnebiMode, pnebiLUT2, d_coefficientsArs2);
    hipMemcpy(coefficientsArs2, d_coefficientsArs2, coeffsMatTotalSz * sizeof (double), hipMemcpyDeviceToHost);
    //end of kernel call for ARS2



    timeStop = std::chrono::system_clock::now();
    double timeArs2 = (double) std::chrono::duration_cast<std::chrono::milliseconds>(timeStop - timeStart).count();
    hipDeviceSynchronize();
    std::cout << "insertIsotropicGaussians() " << timeArs2 << " ms" << std::endl;
    //END OF ARS1




    std::cout << "\nARS Coefficients:\n";
    std::cout << "\ti \tDownward \tLUT\n";
    ars1.setCoefficients(coeffsArs1, coeffsMatNumCols);
    //    for (int i = 0; i < coeffsVectorMaxSz; i++) {
    //        std::cout << "ars1coeff_d[" << i << "] " << d_coeffsMat1[i] << std::endl;
    //    }
    ars2.setCoefficients(coefficientsArs2, coeffsMatNumCols);
    for (int i = 0; i < ars1.coefficients().size() && i < ars2.coefficients().size(); ++i) {
        std::cout << "\t" << i << " \t" << ars1.coefficients().at(i) << " \t" << ars2.coefficients().at(i) << "\n";
    }
    std::cout << std::endl;

    std::vector<double> funcFourierRecursDownLUT;
    std::vector<double> funcFourierRecursDown;
    int thnum = 360;
    double dtheta = M_PI / thnum;
    double theta;
    for (int i = 0; i < thnum; ++i) {
        theta = dtheta * i;
        funcFourierRecursDownLUT.push_back(ars1.eval(theta));
        funcFourierRecursDown.push_back(ars2.eval(theta));
    }

    std::cout << "\nBranch and Bound limits:\n";
    int bbnum = 32;
    std::vector<BoundInterval> bbbs(bbnum);
    for (int i = 0; i < bbnum; ++i) {
        bbbs[i].x0 = M_PI * i / bbnum;
        bbbs[i].x1 = M_PI * (i + 1) / bbnum;
        cuars::findLUFourier(ars1.coefficients(), bbbs[i].x0, bbbs[i].x1, bbbs[i].y0, bbbs[i].y1);
        std::cout << i << ": x0 " << RAD2DEG(bbbs[i].x0) << " x1 " << RAD2DEG(bbbs[i].x1) << ", y0 " << bbbs[i].y0 << " y1 " << bbbs[i].y1 << std::endl;
    }


    cuars::FourierOptimizerBB1D optim(ars1.coefficients());
    double xopt, ymin, ymax;
    optim.enableXTolerance(true);
    optim.enableYTolerance(true);
    optim.setXTolerance(M_PI / 180.0 * 0.5);
    optim.setYTolerance(1.0);
    optim.findGlobalMax(0, M_PI, xopt, ymin, ymax);
    std::cout << "\n****\nMaximum in x = " << xopt << " (" << RAD2DEG(xopt) << " deg), maximum between [" << ymin << "," << ymax << "]" << std::endl;

    double xopt2, ymax2;
    cuars::findGlobalMaxBBFourier(ars1.coefficients(), 0, M_PI, M_PI / 180.0 * 0.5, 1.0, xopt2, ymax2);
    std::cout << "  repeated evaluation with findGlobalMaxBBFourier(): maximum in x " << xopt2 << " (" << RAD2DEG(xopt2) << " deg), maximum value " << ymax2 << std::endl;



    //    //Free GPU and CPU memory
    //    hipFree(d_coefficientsArs2);
    //    hipFree(kernelInput2);
    //    //    free(coefficientsArs2); //array
    //    hipFree(coeffsMat1);
    //    hipFree(kernelInput1);
    //    //    free(coeffsArs1);

    return 0;
}

void rangeToPoint(double* ranges, int num, int numPadded, double angleMin, double angleRes, std::vector<cuars::Vec2d>& points) {
    cuars::Vec2d p;
    for (int i = 0; i < numPadded; ++i) {
        if (i < num) {
            double a = angleMin + angleRes * i;
            p.x = ranges[i] * cos(a);
            p.y = ranges[i] * sin(a);
            points.push_back(p);
        } else {
            //padding with zeros

            p.x = 0.0;
            p.y = 0.0;
            points.push_back(p);
        }
        //        std::cout << p.x << " " << p.y << std::endl;
    }
}

int ceilPow2(int n) {
    ARS_ASSERT(n > 0);

    int exponent = ceil(log2(n));

    int nPadded = std::pow<int>(2, exponent);
    std::cout << "Number of points: " << n << " -> afeer padding = " << nPadded << std::endl;



    return nPadded;
}


