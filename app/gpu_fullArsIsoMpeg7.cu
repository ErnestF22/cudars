#include "hip/hip_runtime.h"

/**
 * ARS - Angular Radon Spectrum 
 * Copyright (C) 2017 Dario Lodi Rizzini.
 *
 * ARS is free software: you can redistribute it and/or modify
 * it under the terms of the GNU Lesser General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 * 
 * ARS is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU Lesser General Public License for more details.
 *
 * You should have received a copy of the GNU Lesser General Public License
 * along with ARS.  If not, see <http://www.gnu.org/licenses/>.
 */


#include <iostream>
#include <chrono>

#include <unordered_map>

#include "ars/Profiler.h"

#include "ars/mpeg7RW.h"
#include "ars/mpeg7_io.h"
#include "ars/cuArsIso.cuh"




#define PRINT_DIM(X) std::cout << #X << " rows " << X.rows() << " cols " << X.cols() << std::endl;
#define RAD2DEG(X) (180.0/M_PI*(X))




void findComparisonPair(const std::vector<std::string>& inputFilenames, std::vector<std::pair<int, int> >& comPairs);

void filterComparisonPair(std::string resumeFilename, std::ostream& outputfile,
        const std::vector<std::string>& inputFilenames, std::vector<std::pair<int, int> >& inputPairs,
        std::vector<std::pair<int, int> >& outputPairs);

std::string getPrefix(std::string filename);

std::string getShortName(std::string filename);

std::string getLeafDirectory(std::string filename);

double mod180(double angle);

struct BoundInterval {
    double x0;
    double x1;
    double y0;
    double y1;
};

int main(int argc, char **argv) {
    cuars::AngularRadonSpectrum2d arsSrc;
    cuars::AngularRadonSpectrum2d arsDst;
    ArsImgTests::PointReaderWriter pointsSrc;
    ArsImgTests::PointReaderWriter pointsDst;


    rofl::ParamMap params;
    std::string filenameCfg;
    std::string filenameSrc;
    std::string filenameDst;
    std::string filenameRot;
    std::string filenameArsSrc;
    std::string filenameArsDst;
    std::string filenameArsRot;
    std::string filenameArsCor;
    std::string filenameCovSrc;
    std::string filenameCovDst;
    int arsOrder;
    double arsSigma, arsThetaToll;
    double rotTrue, rotArs;
    //The variables below are for I/O related functionalities (plot, etc.) that are highly Eigen-based and are present in the CPU-only ArsImgTests...
    //Maybe implement them later
    //    double sampleRes, sampleAng; 
    //    int sampleNum;
    //    bool saveOn;
    //    bool saveCov;


    params.read(argc, argv);
    params.getParam<std::string>("cfg", filenameCfg, "");
    std::cout << "config filename: " << filenameCfg << std::endl;
    if (filenameCfg != "") {
        params.read(filenameCfg);
    }

    params.read(argc, argv);
    params.getParam<std::string>("src", filenameSrc, "/home/rimlab/Downloads/mpeg7_point_tests/noise000_occl00_rand000/apple-1_xp0686_yp0967_t059_sigma0001_occl000.txt");
    params.getParam<std::string>("dst", filenameDst, "/home/rimlab/Downloads/mpeg7_point_tests/noise000_occl00_rand000/apple-1_xp0749_yn0521_t090_sigma0001_occl000.txt");
    params.getParam<int>("arsOrder", arsOrder, 20);
    params.getParam<double>("arsSigma", arsSigma, 1.0);
    params.getParam<double>("arsTollDeg", arsThetaToll, 1.0);
    arsThetaToll *= M_PI / 180.0;
    //    params.getParam<double>("sampleResDeg", sampleRes, 0.5);
    //    sampleRes *= M_PI / 180.0;
    //    params.getParam<bool>("saveOn", saveOn, false);
    //    params.getParam<bool>("saveCov", saveCov, false);

    std::cout << "\nParameter values:\n";
    params.write(std::cout);
    std::cout << std::endl;


    /* Reading files from folder */
    std::string inputGlob;
    std::vector<std::string> inputFilenames;

    mpeg7io::getDirectoryFiles(inputGlob, inputFilenames);
    std::cout << "\nFilenames:\n";
    size_t numFiles = 0;
    for (auto& filename : inputFilenames) {
        if (numFiles < 30)
            std::cout << "  " << getPrefix(filename) << " " << getShortName(filename) << " " << filename << "\n";
        else if (numFiles == 30)
            std::cout << "..." << std::endl;

        numFiles++;
    }
    std::cout << std::endl;




    //ARS parameters setting
    arsSrc.setARSFOrder(arsOrder);
    arsDst.setARSFOrder(arsOrder);
    cuars::ArsKernelIsotropic2d::ComputeMode pnebiMode = cuars::ArsKernelIsotropic2d::ComputeMode::PNEBI_DOWNWARD;
    arsSrc.setComputeMode(pnebiMode);
    arsDst.setComputeMode(pnebiMode);


    //Fourier coefficients mega-matrix computation -> parallelization parameters
    int numPts = std::min<int>(pointsSrc.points().size(), pointsDst.points().size()); //the two should normally be equal
    const int numPtsAfterPadding = ceilPow2(numPts); //for apple1 -> numPts 661; padded 1024
    const int blockSize = 256; //num threads per block
    const int numBlocks = (numPtsAfterPadding * numPtsAfterPadding) / blockSize; //number of blocks in grid (each block contains blockSize threads)
    const int gridTotalSize = blockSize*numBlocks; //total number of threads in grid
    //depth of mega-matrix
    const int coeffsMatNumCols = 2 * arsOrder + 2;
    const int coeffsMatNumColsPadded = ceilPow2(coeffsMatNumCols);
    const int coeffsMatTotalSz = numPtsAfterPadding * numPtsAfterPadding * coeffsMatNumColsPadded;
    //Fourier matrix sum -> parallelization parameters
    const int sumBlockSz = 64;
    const int sumGridSz = 256; //can be used to futher parallelize sum of mega-matrix (for now in sum kernel it is actually set to 1)
    std::cout << "Parallelization params:" << std::endl;
    std::cout << "numPtsAfterPadding " << numPtsAfterPadding << " blockSize " << blockSize << " numBlocks " << numBlocks << " gridTotalSize " << gridTotalSize << std::endl;
    std::cout << "sumBlockSz " << sumBlockSz << " sumGridSz " << sumGridSz << std::endl;

    std::cout << "\n------\n" << std::endl;

    std::cout << "\n\nCalling kernel functions on GPU\n" << std::endl;


    //ARS SRC -> preparation for kernel calls and kernel calls
    hipEvent_t startSrc, stopSrc; //timing using CUDA events
    hipEventCreate(&startSrc);
    hipEventCreate(&stopSrc);
    cuars::Vec2d * kernelInputSrc;
    hipMalloc((void**) &kernelInputSrc, numPtsAfterPadding * sizeof (cuars::Vec2d));
    hipMemcpy(kernelInputSrc, pointsSrc.points().data(), numPtsAfterPadding * sizeof (cuars::Vec2d), hipMemcpyHostToDevice);

    double *coeffsMatSrc;
    hipMalloc((void**) &coeffsMatSrc, coeffsMatTotalSz * sizeof (double));
    hipMemset(coeffsMatSrc, 0.0, coeffsMatTotalSz * sizeof (double));
    //    for (int i = 0; i < coeffsMatTotalSz; ++i) {
    //        coeffsMaSrc1[i] = 0.0;
    //    }
    double* d_coeffsArsSrc;
    hipMalloc((void**) &d_coeffsArsSrc, coeffsMatNumColsPadded * sizeof (double));
    hipMemset(d_coeffsArsSrc, 0.0, coeffsMatNumColsPadded * sizeof (double));

    hipEventRecord(startSrc);
    iigKernelDownward << <numBlocks, blockSize >> >(kernelInputSrc, arsSigma, arsSigma, numPts, numPtsAfterPadding, arsOrder, coeffsMatNumColsPadded, pnebiMode, coeffsMatSrc);
    sumColumns << <1, sumBlockSz>> >(coeffsMatSrc, numPtsAfterPadding, coeffsMatNumColsPadded, d_coeffsArsSrc);
    hipEventRecord(stopSrc);

    double* coeffsArsSrc = new double [coeffsMatNumColsPadded];
    hipMemcpy(coeffsArsSrc, d_coeffsArsSrc, coeffsMatNumColsPadded * sizeof (double), hipMemcpyDeviceToHost);

    hipEventSynchronize(stopSrc);
    float millisecondsSrc = 0.0f;
    hipEventElapsedTime(&millisecondsSrc, startSrc, stopSrc);
    std::cout << "SRC -> insertIsotropicGaussians() " << millisecondsSrc << " ms" << std::endl;

    hipError_t cudaerr = hipDeviceSynchronize();
    if (cudaerr != hipSuccess)
        printf("kernel launch failed with error \"%s\".\n", hipGetErrorString(cudaerr));

    //    for (int i = 0; i < coeffsMatNumColsPadded; ++i) {
    //        std::cout << "coeffsArsSrc[" << i << "] " << coeffsArsSrc[i] << std::endl;
    //    }

    hipFree(coeffsMatSrc);
    hipFree(kernelInputSrc);
    hipFree(d_coeffsArsSrc);
    hipEventDestroy(startSrc);
    hipEventDestroy(stopSrc);
    //END OF ARS SRC



    std::cout << "\n------\n" << std::endl; //"pause" between ars src and ars dst



    //ARS DST -> preparation for kernel calls and kernel calls
    hipEvent_t startDst, stopDst; //timing using CUDA events
    hipEventCreate(&startDst);
    hipEventCreate(&stopDst);
    cuars::Vec2d *kernelInputDst;
    hipMalloc((void**) &kernelInputDst, numPtsAfterPadding * sizeof (cuars::Vec2d));
    hipMemcpy(kernelInputDst, pointsDst.points().data(), numPtsAfterPadding * sizeof (cuars::Vec2d), hipMemcpyHostToDevice);

    double *coeffsMatDst; //magari evitare di fare il delete e poi riallocarla è più efficiente (anche se comunque ci sarebbe poi da settare tutto a 0)
    hipMalloc((void**) &coeffsMatDst, coeffsMatTotalSz * sizeof (double));
    hipMemset(coeffsMatDst, 0.0, coeffsMatTotalSz * sizeof (double));
    //    for (int i = 0; i < coeffsMatTotalSz; ++i) {
    //        coeffsMatDst[i] = 0.0;
    //    }
    double* d_coeffsArsDst;
    hipMalloc((void**) &d_coeffsArsDst, coeffsMatNumColsPadded * sizeof (double));
    hipMemset(d_coeffsArsDst, 0.0, coeffsMatNumColsPadded * sizeof (double));

    hipEventRecord(startDst);
    iigKernelDownward << <numBlocks, blockSize >> >(kernelInputDst, arsSigma, arsSigma, numPts, numPtsAfterPadding, arsOrder, coeffsMatNumColsPadded, pnebiMode, coeffsMatDst);
    sumColumns << <1, sumBlockSz>> >(coeffsMatDst, numPtsAfterPadding, coeffsMatNumColsPadded, d_coeffsArsDst);
    hipEventRecord(stopDst);



    double* coeffsArsDst = new double [coeffsMatNumColsPadded];
    hipMemcpy(coeffsArsDst, d_coeffsArsDst, coeffsMatNumColsPadded * sizeof (double), hipMemcpyDeviceToHost);

    hipEventSynchronize(stopDst);
    float millisecondsDst = 0.0f;
    hipEventElapsedTime(&millisecondsDst, startDst, stopDst);
    std::cout << "DST -> insertIsotropicGaussiansDst() " << millisecondsDst << " ms" << std::endl;

    cudaerr = hipDeviceSynchronize();
    if (cudaerr != hipSuccess)
        printf("kernel launch failed with error \"%s\".\n", hipGetErrorString(cudaerr));

    //    for (int i = 0; i < coeffsMatNumColsPadded; ++i) {
    //        std::cout << "coeffsArsDst[" << i << "] " << coeffsArsDst[i] << std::endl;
    //    }

    hipFree(coeffsMatDst);
    hipFree(kernelInputDst);
    hipFree(d_coeffsArsDst);
    hipEventDestroy(startDst);
    hipEventDestroy(stopDst);
    //END OF ARS DST





    //Computation final computations (correlation, ...) on CPU
    std::cout << "\nARS Coefficients:\n";
    std::cout << "Coefficients: Src, Dst, Cor" << std::endl;

    double thetaMax, corrMax, fourierTol;
    fourierTol = 1.0; // TODO: check for a proper tolerance

    std::vector<double> coeffsCor;
    {
        cuars::ScopedTimer("ars.correlation()");
        std::vector<double> tmpSrc;
        tmpSrc.assign(coeffsArsSrc, coeffsArsSrc + coeffsMatNumColsPadded);
        std::vector<double> tmpDst;
        tmpDst.assign(coeffsArsDst, coeffsArsDst + coeffsMatNumColsPadded);
        cuars::computeFourierCorr(tmpSrc, tmpDst, coeffsCor);
        cuars::findGlobalMaxBBFourier(coeffsCor, 0.0, M_PI, arsThetaToll, fourierTol, thetaMax, corrMax);
        rotArs = thetaMax;
    }



    arsSrc.setCoefficients(coeffsArsSrc, coeffsMatNumCols);
    //    for (int i = 0; i < coeffsVectorMaxSz; i++) {
    //        std::cout << "arsSrc - coeff_d[" << i << "] " << d_coeffsMat1[i] << std::endl;
    //    }
    arsDst.setCoefficients(coeffsArsDst, coeffsMatNumCols);
    for (int i = 0; i < arsSrc.coefficients().size() && i < arsDst.coefficients().size(); ++i) {
        std::cout << "\t" << i << " \t" << arsSrc.coefficients().at(i) << " \t" << arsDst.coefficients().at(i) << " \t" << coeffsCor[i] << std::endl;
    }
    std::cout << std::endl;



    // Computes the rotated points,centroid, affine transf matrix between src and dst
    ArsImgTests::PointReaderWriter pointsRot(pointsSrc.points());
    cuars::Vec2d centroidSrc = pointsSrc.computeCentroid();
    cuars::Vec2d centroidDst = pointsDst.computeCentroid();
    cuars::Affine2d rotSrcDst = ArsImgTests::PointReaderWriter::coordToTransform(0.0, 0.0, rotArs);
    //    cuars::Vec2d translSrcDst = centroidDst - rotSrcDst * centroidSrc;
    cuars::Vec2d translSrcDst;
    cuars::vec2diff(translSrcDst, centroidDst, cuars::aff2TimesVec2WRV(rotSrcDst, centroidSrc));
    //    std::cout << "centroidSrc " << centroidSrc.transpose() << "\n"
    //            << "rotSrcDst\n" << rotSrcDst.matrix() << "\n"
    //            << "translation: [" << translSrcDst.transpose() << "] rotation[deg] " << (180.0 / M_PI * rotArs) << "\n";
    std::cout << "centroidSrc " << centroidSrc.x << " \t" << centroidSrc.y << "\n"
            << "centroidDst " << centroidDst.x << " \t" << centroidDst.y << "\n"
            << "rotSrcDst\n" << rotSrcDst << "\n"
            << "translation: [" << translSrcDst.x << " \t" << translSrcDst.y << "] rotation[deg] " << (180.0 / M_PI * rotArs) << "\n";
    pointsRot.applyTransform(translSrcDst.x, translSrcDst.y, rotArs);



    rotTrue = pointsDst.getRotTheta() - pointsSrc.getRotTheta();
    std::cout << "\n***\npointsDst.getrotTheta() [deg]" << (180 / M_PI * pointsDst.getRotTheta())
            << ", pointsSrc.getrotTheta() [deg] " << (180.0 / M_PI * pointsSrc.getRotTheta()) << "\n";
    std::cout << "rotTrue[deg] \t" << (180.0 / M_PI * rotTrue) << " \t" << (180.0 / M_PI * mod180(rotTrue)) << std::endl;
    std::cout << "rotArs[deg] \t" << (180.0 / M_PI * rotArs) << " \t" << (180.0 / M_PI * mod180(rotArs)) << std::endl;

    //Free CPU memory
    free(coeffsArsSrc);
    free(coeffsArsDst);


    return 0;
}

// Reads outputFilename for the list of already processed files

void filterComparisonPair(std::string resumeFilename, std::ostream& outputFile,
        const std::vector<std::string>& inputFilenames, std::vector<std::pair<int, int> >& inputPairs,
        std::vector<std::pair<int, int> >& outputPairs) {
    std::unordered_multimap<std::string, int> indicesMap;
    std::vector<std::pair<int, int> > visitedPairs;
    std::string filenameShort, line, label1, label2;
    int numIn1, numOccl1, numRand1, i1, i2;

    outputPairs.clear();
    // Visits all the lines/items of the output file
    for (int i = 0; i < inputFilenames.size(); ++i) {
        filenameShort = getShortName(inputFilenames[i]);
        indicesMap.insert(std::make_pair(filenameShort, i));
    }

    // Finds all the pairs already visited
    std::ifstream resumeFile(resumeFilename.c_str());
    if (!resumeFile) {
        std::cerr << "Cannot open file \"" << resumeFilename << "\": nothing to resume" << std::endl;
        outputPairs.insert(outputPairs.begin(), inputPairs.begin(), inputPairs.end());
        return;
    }
    while (!resumeFile.eof()) {
        std::getline(resumeFile, line);
        outputFile << line << "\n";
        // Strips comment from line
        size_t pos = line.find_first_of('#');
        if (pos != std::string::npos) {
            line = line.substr(0, pos);
        }
        // Reads the labels of the two files from items 
        std::stringstream ssline(line);
        if (ssline >> label1 >> numIn1 >> numOccl1 >> numRand1 >> label2) {
            // Finds the indices of label1 and label2
            auto iter1 = indicesMap.find(label1);
            if (iter1 == indicesMap.end()) i1 = -1;
            else i1 = iter1->second;
            auto iter2 = indicesMap.find(label2);
            if (iter2 == indicesMap.end()) i2 = -1;
            else i2 = iter2->second;
            std::cout << "  visited \"" << label1 << "\" [" << i1 << "] \"" << label2 << "\" [" << i2 << "]\n";
            // If both labels are found, it inserts the pair
            if (i1 >= 0 && i2 >= 0) {
                if (i1 != i2) {
                    visitedPairs.push_back(std::make_pair(i1, i2));
                } else {
                    // two files with the same short name are handled...
                    std::cout << "  homonymous \"" << label1 << "\": ";
                    auto range = indicesMap.equal_range(label1);
                    for (iter1 = range.first; iter1 != range.second; ++iter1) {
                        iter2 = iter1;
                        std::advance(iter2, 1);
                        for (; iter2 != range.second; ++iter2) {
                            i1 = iter1->second;
                            i2 = iter2->second;
                            if (i1 > i2) std::swap(i1, i2);
                            visitedPairs.push_back(std::make_pair(i1, i2));
                            std::cout << " (" << i1 << "," << i2 << ") ";
                        }
                    }
                    std::cout << std::endl;
                }
            }
        }
    }
    resumeFile.close();
    outputFile << "# RESUMING" << std::endl;

    // Finds the set difference
    std::sort(inputPairs.begin(), inputPairs.end());
    std::sort(visitedPairs.begin(), visitedPairs.end());
    std::set_difference(inputPairs.begin(), inputPairs.end(),
            visitedPairs.begin(), visitedPairs.end(),
            std::back_inserter(outputPairs));

    std::cout << "Remaining pairs:\n";
    for (auto& p : outputPairs) {
        std::cout << " " << p.first << ", " << p.second << ": \"" << getShortName(inputFilenames[p.first]) << "\", \"" << getShortName(inputFilenames[p.second]) << "\"\n";
    }
    std::cout << "All pairs " << inputPairs.size() << ", visited pairs " << visitedPairs.size() << ", remaining pairs " << outputPairs.size() << std::endl;
}

std::string getPrefix(std::string filename) {
    // Strips filename of the path 
    std::experimental::filesystem::path filepath(filename);
    std::string name = filepath.filename().string();
    std::string prefix;
    //  std::cout << "  name: \"" << name << "\"\n";

    // Finds the prefix
    size_t pos = name.find_first_of('_');
    if (pos != std::string::npos) {
        prefix = name.substr(0, pos);
    } else {
        prefix = name;
    }
    return prefix;
}

std::string getShortName(std::string filename) {
    std::stringstream ss;
    std::string prefix = getPrefix(filename);
    std::experimental::filesystem::path filenamePath = filename;
    filename = filenamePath.filename().string();
    // Computes a digest on the string
    unsigned int h = 19;
    for (int i = 0; i < filename.length(); ++i) {
        h = ((h * 31) + (unsigned int) filename[i]) % 97;
    }
    //  std::cout << "\nglob \"" << filenamePath.string() << "\" filename \"" << filename << "\" hash " << h << std::endl;
    ss << prefix << "_" << std::setw(2) << std::setfill('0') << h;
    return ss.str();
}

std::string getLeafDirectory(std::string filename) {
    std::experimental::filesystem::path filenamePath = filename;
    std::string parent = filenamePath.parent_path().string();
    size_t pos = parent.find_last_of('/');
    std::string leafDir = "";
    if (pos != std::string::npos) {
        leafDir = parent.substr(pos + 1, parent.length());
    }
    return leafDir;
}

double mod180(double angle) {
    return (angle - floor(angle / M_PI) * M_PI);
}
