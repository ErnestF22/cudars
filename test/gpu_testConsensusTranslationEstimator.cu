#include "hip/hip_runtime.h"
#include <iostream>
#include "ars/utils.h"
#include "ars/ars2d.cuh"
#include <ars/ars2d.h>
#include <ars/ConsensusTranslationEstimator.cuh>
#include <rofl/common/param_map.h>

int main(int argc, char **argv)
{
    // cuars::ConsensusTranslationEstimator2d translEstim;
    cuars::VecVec2d pointsSrc, pointsDst, translCandidates;
    rofl::ParamMap params;
    std::string filenameCfg;

    cuars::ArsTec2dParams translParams;
    // ParlArsIsoParams paiParams;

    // Reads params from command line
    params.read(argc, argv);
    params.getParam("cfg", filenameCfg, std::string(""));
    params.read(filenameCfg);
    params.read(argc, argv);
    params.getParam<double>("translRes", translParams.translRes, 1.0);
    // params.getParamContainer("translMin", translMin.data(), translMin.data() + translMin.size(), "[-10.0,-10.0]", double(0.0), "[,]"); //TODO: adapt ParamContainer to Cuda types
    params.getParam<double>("translMin-x", translParams.translMin.x, -10.0);
    params.getParam<double>("translMin-y", translParams.translMin.y, -10.0);
    // params.getParamContainer("translGt", translGt.data(), translGt.data() + translGt.size(), "[-4.2,5.0]", double(1.0), "[,]");
    params.getParam<double>("translGt-x", translParams.translGt.x, -4.2);
    params.getParam<double>("translGt-y", translParams.translGt.y, 5.0);
    params.getParamContainer("gridSize", translParams.gridSize.data(), translParams.gridSize.data() + translParams.gridSize.size(), "[21,21]", int(0), "[,]");
    params.getParamContainer("gridWin", translParams.gridWin.data(), translParams.gridWin.data() + translParams.gridWin.size(), "[1,1]", int(1), "[,]");
    params.getParam<bool>("adaptive", translParams.adaptiveGrid, false);
    params.getParam<bool>("plot", translParams.plot, false);

    std::cout << "\nParams:" << std::endl;
    params.write(std::cout);
    std::cout << "-------\n"
              << std::endl;

    for (int i = 0; i < 10; ++i)
    {
        cuars::Vec2d p;
        cuars::fillVec2d(p, (1.0 + 0.4 * i), (-2.0 - 0.35 * i));
        pointsSrc.push_back(p);
        pointsDst.push_back(cuars::vec2sumWRV(p, translParams.translGt));
    }
    pointsDst.push_back(make_double2(0.0, 0.0));
    pointsDst.push_back(make_double2(4.0, 4.0));
    pointsSrc.push_back(make_double2(3.2, 6.2));
    pointsSrc.push_back(make_double2(3.5, 2.6));

    std::cout << "Source point set:\n";
    for (auto &pt : pointsSrc)
    {
        std::cout << "  [";
        // cuars::printVec2d(pt);
        std::cout << pt.x << "\t" << pt.y;
        std::cout << "]\n";
    }
    std::cout << "Destination point set:\n";
    for (auto &pt : pointsDst)
    {
        std::cout << "  [";
        // cuars::printVec2d(pt);
        std::cout << pt.x << "\t" << pt.y;
        std::cout << "]\n";
    }

    cuars::computeArsTec2d(translCandidates, pointsSrc, pointsDst, translParams);

    std::cout << "Estimated translation values:\n";
    // cuars::ConsensusTranslationEstimator2d translEstimOutput(...) //constructor can be used for example to fill the class with the outputs
    for (auto &pt : translCandidates)
    {
        std::cout << "  [";
        // cuars::printVec2d(pt);
        std::cout << pt.x << "\t" << pt.y;
        std::cout << "]\n";
    }

    return 0;
}
